#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "error.hpp"

template<unsigned int nVars>
__global__
void interp_u(const double* const*  U_spts, double *U_out,
    const int* __restrict__ donors, const double* __restrict__ weights,
    const char* __restrict__ etypes, const int* __restrict__ wgt_inds,
    const int* __restrict__ out_inds, int nFringe,
    const int* __restrict__ nweights, const int* __restrict__ strides)
{
  const int fpt = blockDim.x * blockIdx.x + threadIdx.x;

  if (fpt >= nFringe)
    return;
  
  // modification stides[4*ntypes]

  const int ind = nVars * out_inds[fpt];
  const int N = (int)etypes[fpt];
  //const int u_ind = donors[fpt] * strides[3*N+0];
  //const int u_ind = donors[fpt] * strides[4*N+0];
  // different RAM partern
  const int soasz = strides[4*N+3];
  const int u_ind = (donors[fpt]/soasz)*strides[4*N+0]+donors[fpt]%soasz;
  const int w_ind = wgt_inds[fpt];
  const int nSpts = nweights[fpt];
  //const int sstride = strides[3*N+1];
  //const int vstride = strides[3*N+2];
  const int sstride = strides[4*N+1];
  const int vstride = strides[4*N+2];

  double sum[nVars] = {0.0};

  for (int spt = 0; spt < nSpts; spt++)
  {
    double wt = weights[w_ind+spt];
    for (int var = 0; var < nVars; var++)
      sum[var] += wt * U_spts[N][u_ind + spt*sstride + var*vstride];
  }

  for (int var = 0; var < nVars; var++) {
    U_out[ind+var] = sum[var];
    /*
    if(sum[var]>200.0) {
        for(int spt=0;spt<nSpts;spt++) {
            printf(" %15.7e ", U_spts[N][u_ind + spt*sstride + var*vstride] );
        }
        printf(" \n");
    }
    */
  }

}

void interp_u_wrapper(double **U_spts, double *U_out, int *donors,
    double *weights, char *etypes, int* wgt_inds, int* out_inds, int nFringe,
    int* nSpts, int nVars, int *strides, hipStream_t stream_h)
{
  unsigned int threads = 128;
  unsigned int blocks = (nFringe + threads - 1) / threads;

  if (nVars == 1)
    interp_u<1><<<blocks, threads, 0, stream_h>>>(U_spts, U_out, donors, weights, etypes,
        wgt_inds, out_inds, nFringe, nSpts, strides);
  else if (nVars == 4)
    interp_u<4><<<blocks, threads, 0, stream_h>>>(U_spts, U_out, donors, weights, etypes,
        wgt_inds, out_inds, nFringe, nSpts, strides);
  else if (nVars == 5)
    interp_u<5><<<blocks, threads, 0, stream_h>>>(U_spts, U_out, donors, weights, etypes,
        wgt_inds, out_inds, nFringe, nSpts, strides);

  check_error();
}


template <unsigned int nDims, unsigned int nVars>
__global__
void interp_du(const double* const*  dU_spts, double *dU_out,
    const int* __restrict__ donors, const double* __restrict__ weights,
    const char* __restrict__ etypes, const int* __restrict__ wgt_inds,
    const int* __restrict__ out_inds, int nFringe,
    const int* __restrict__ nweights, const int* __restrict__ strides)
{
  const int fpt = blockDim.x * blockIdx.x + threadIdx.x;

  if (fpt >= nFringe)
    return;

  // to be consistent with python ram layout
  const int N = (int)etypes[fpt];
  const int soasz = strides[5*N+4];
  const int u_ind = (donors[fpt]/soasz)*strides[5*N+0]+donors[fpt]%soasz;
  //const int u_ind = donors[fpt] * strides[4*N+0];
  const int w_ind = wgt_inds[fpt];
  const int nSpts = nweights[fpt];
  //const int sstride = strides[4*N+1];
  //const int vstride = strides[4*N+2];
  //const int dstride = strides[4*N+3];
  const int sstride = strides[5*N+1];
  const int vstride = strides[5*N+2];
  const int dstride = strides[5*N+3];

  double sum[nDims][nVars] = {0.0};

  for (int spt = 0; spt < nSpts; spt++)
  {
    double wgt = weights[w_ind + spt];
    for (int dim = 0; dim < nDims; dim++)
      for (int var = 0; var < nVars; var++)
        sum[dim][var] += wgt * dU_spts[N][u_ind + spt*sstride + dim*dstride + var * vstride];
  }

  for (int dim = 0; dim < nDims; dim++)
  {
    for (int var = 0; var < nVars; var++)
    {
      int ind = var + nVars * (dim + nDims * out_inds[fpt]);
      dU_out[ind] = sum[dim][var];
    }
  }
}

void interp_du_wrapper(double **dU_spts, double *dU_out, int *donors,
    double *weights, char *etypes, int* wgt_inds, int* out_inds, int nFringe,
    int* nSpts, int nVars, int nDims, int *strides, hipStream_t stream_h)
{
  int threads = 128;
  int blocks = (nFringe + threads - 1) / threads;

  if (nDims == 3)
  {
    if (nVars == 1)
      interp_du<3,1><<<blocks, threads, 0, stream_h>>>(dU_spts, dU_out, donors,
          weights, etypes, wgt_inds, out_inds, nFringe, nSpts, strides);
    else if (nVars == 5)
      interp_du<3,5><<<blocks, threads, 0, stream_h>>>(dU_spts, dU_out, donors,
          weights, etypes, wgt_inds, out_inds, nFringe, nSpts, strides);
    else
      FatalError("3D nVars case not recognized (expecting 1 or 5)");
  }
  else
    FatalError("TIOGA support for 3D only currently!");

  check_error();
}
