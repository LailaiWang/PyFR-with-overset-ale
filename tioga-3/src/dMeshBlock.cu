#include "hip/hip_runtime.h"
#include "dMeshBlock.h"
#include "funcs.hpp"

#include "hip/hip_runtime.h"
#include "math.h"

#define MAX_UCHAR 255

#define FULL_MASK 0xffffffff

/* --- Handy Vector Operation Macros --- */

#define NF1 16 // 16-32 depending on unstructured-ness of grid & desire for robustness
#define NF2  4 // 3-6 depending on unstructured-ness of grid & desire for robustness

#define CROSS(a, b, c) { \
  c[0] = a[1]*b[2] - a[2]*b[1]; \
  c[1] = a[2]*b[0] - a[0]*b[2]; \
  c[2] = a[0]*b[1] - a[1]*b[0]; }

#define CROSS4(a1, a2, b1, b2, c) { \
  c[0] = (a1[1]-a2[1])*(b1[2]-b2[2]) - (a1[2]-a2[2])*(b1[1]-b2[1]); \
  c[1] = (a1[2]-a2[2])*(b1[0]-b2[0]) - (a1[0]-a2[0])*(b1[2]-b2[2]); \
  c[2] = (a1[0]-a2[0])*(b1[1]-b2[1]) - (a1[1]-a2[1])*(b1[0]-b2[0]); }

#define DOT(a, b) (a[0]*b[0] + a[1]*b[1] + a[2]*b[2])

#define NORM(a) sqrt(a[0]*a[0]+a[1]*a[1]+a[2]*a[2])

static
__device__ __forceinline__
float DOTCROSS4(const float* __restrict__ c,
                 const float* __restrict__ a1, const float* __restrict__ a2,
                 const float* __restrict__ b1, const float* __restrict__ b2)
{
  float d[3];
  CROSS4(a1,a2,b1,b2,d)
  return DOT(c,d);
}


/* --- Misc. Helpful CUDA kernels --- */
__device__ __forceinline__
void print_nodes(const double* pts, int id, int npts)
{
  int idx = threadIdx.x;
  for (int tid = 0; tid < 32; tid++)
  {
    if (idx == tid)
    {
      printf("Points%d = [",id);
      for (int i = 0; i < npts - 1; i++)
        printf("%f %f %f;\n",pts[3*i+0],pts[3*i+1],pts[3*i+2]);

      int I = npts-1;
      printf("%f %f %f];\n",pts[3*I+0],pts[3*I+1],pts[3*I+2]);
    }
  }
}

__device__ __forceinline__
void print_nodes(const float* pts, int id, int npts)
{
  int idx = threadIdx.x;
  for (int tid = 0; tid < 32; tid++)
  {
    if (idx == tid)
    {
      printf("Points%d = [",id);
      for (int i = 0; i < npts - 1; i++)
        printf("%f %f %f;\n",pts[3*i+0],pts[3*i+1],pts[3*i+2]);

      int I = npts-1;
      printf("%f %f %f];\n",pts[3*I+0],pts[3*I+1],pts[3*I+2]);
    }
  }
}

#define WARP_SZ 32

__device__
inline int lane_id(void) { return threadIdx.x % WARP_SZ; }

__device__
inline int warp_bcast(unsigned int mask, int v, int leader) { return __shfl_sync(mask, v, leader); }

__device__ __forceinline__
float warpAllReduceMin(float val)
{
  for (int mask = warpSize/2; mask > 0; mask /= 2)
    val = fminf(val, __shfl_xor_sync(FULL_MASK, val, mask));
  return val;
}

__device__
int floatToOrderedInt(float floatVal)
{
  int intVal = __float_as_int(floatVal);

  return (intVal >= 0) ? intVal : intVal ^ 0x8FFFFFFF;
}

__device__
unsigned int floatToUint(float fval)
{
  unsigned int ival = __float_as_uint(fval);
  unsigned int mask = -int(ival >> 31) | 0x80000000;
  return ival ^ mask;
}

__device__
float uintToFloat(unsigned int ival)
{
  unsigned int mask = ((ival >> 31) - 1) | 0x80000000;
  return __uint_as_float(ival ^ mask);
}

__device__
float orderedIntToFloat(int intVal)
{
  return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x8FFFFFFF );
}

__device__ float atomicMaxf(float* address, float val)
{
  int old = __float_as_int(*address);
  int assumed;
  while (val > __int_as_float(old))
  {
    assumed = old;
    old = atomicCAS((int*)address, assumed, __float_as_int(val));
  }
  return __int_as_float(old);
}

__device__ float atomicMinf(float* address, float val)
{
  int old = __float_as_int(*address);
  int assumed;
  while (val < __int_as_float(old))
  {
    assumed = old;
    old = atomicCAS((int*)address, assumed, __float_as_int(val));
  }
  return __int_as_float(old);
}

/*! Warp-aggregated atomic increment
 *  https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/ */
__device__
int atomicAggInc(int *ctr)
{
  int mask = __ballot_sync(FULL_MASK, 1);
  // select the leader
  int leader = __ffs(mask) - 1;
  // leader does the update
  int res;
  if (lane_id() == leader)
    res = atomicAdd(ctr, __popc(mask));
  // brodcast result
  res = warp_bcast(mask, res, leader);
  // each thread computes its own value
  return res + __popc(mask & ((1 << lane_id()) - 1));
}

/* ------ dMeshBlock Member Functions ------ */

void dMeshBlock::dataToDevice(int ndims, int nnodes, int ncells, int ncells_adt,
    int nsearch, int* nv, int* nc, int* eleList, double* eleBBox, int* isearch,
    double* xsearch, int rank)
{
  this->nnodes = nnodes;
  this->ncells = ncells;
  this->nc_adt = ncells_adt;

  this->nv = nv;
  this->nc = nc;

  nvert = nv[0];

  this->rank = rank;

  this->eleBBox.assign(eleBBox, ncells_adt*ndims*2);
  this->eleList.assign(eleList, ncells_adt);

  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);
  
  // need to consider multiple type of elements
  auto ijk2gmsh_h = tg_funcs::structured_to_gmsh_hex(nvert);
  ijk2gmsh.assign(ijk2gmsh_h.data(), ijk2gmsh_h.size());

  int nSide = std::cbrt(nvert);

  std::vector<double> xlist_h(nSide);
  std::vector<float> xlistf_h(nSide);
  double dxi = 2./(nSide-1);

  for (int i = 0; i < nSide; i++)
    xlist_h[i] = -1. + i*dxi;

  for (int i = 0; i < nSide; i++)
    xlistf_h[i] = xlist_h[i];

  xlist.assign(xlist_h.data(), xlist_h.size());
  xlistf.assign(xlistf_h.data(), xlistf_h.size());
}

void dMeshBlock::extraDataToDevice(int* vconn)
{
//  c2v.assign(vconn, nvert*ncells);
}

void dMeshBlock::assignHoleMap(bool hasWall, int* nx, int* sam, double* extents)
{
  if (hasWall)
  {
    int size = nx[0]*nx[1]*nx[2];

    std::vector<char> tmp_sam(size);
    for (int i = 0; i < size; i++)
      tmp_sam[i] = (char)sam[i];

    double dx[3];
    for (int d = 0; d < 3; d++)
      dx[d] = (extents[d+3] - extents[d]) / nx[d];

    hm_sam.assign(tmp_sam.data(), size);
    hm_extents.assign(extents, 6);
    hm_nx.assign(nx, 3);
    hm_dx.assign(dx, 3);
  }
  else
  {
    clearHoleMap();
  }
}

void dMeshBlock::clearHoleMap(void)
{
  int nx[3] = {0,0,0};
  double dx[3] = {0,0,0};
  double extents[6] = {0,0,0,0,0,0};

  hm_sam.resize(0);

  hm_nx.assign(nx, 3);
  hm_dx.assign(dx, 3);
  hm_extents.assign(extents, 6);
}

void dMeshBlock::updateADTData(int ncells_adt, int* eleList, double* eleBBox)
{
  this->eleBBox.assign(eleBBox, ncells_adt*nDims*2);
  this->eleList.assign(eleList, ncells_adt);
}

void dMeshBlock::updateSearchPoints(int nsearch, int *isearch, double *xsearch)
{
  this->nsearch = nsearch;
  this->isearch.assign(isearch, nsearch);
  this->xsearch.assign(xsearch, nsearch*nDims);
  rst.resize(nsearch*nDims);
  donorId.resize(nsearch);
}

void dMeshBlock::setDeviceData(double* vx, double* ex, int* ibc, int* ibf)
{
  x = vx;
  iblank_cell = ibc;
  iblank_face = ibf;
  coord = ex;
}

void dMeshBlock::setTransform(double* mat, double* off, int ndim)
{
  if (ndim != nDims)
    ThrowException("dMeshBlock::set_transform: input ndim != nDims");

  rrot = true; /// WORKING ON ADT REBUILD - DISABLED RROT
  Rmat.assign(mat, ndim*ndim);
  offset.assign(off, ndim);
}

/* ---------------------------- Geometric Functions --------------------------- */

__device__ __forceinline__
int oobbCheck(float* vA, float* xA, float* dxA, float* vB, float* xB, float* dxB)
{
  double eps = 1e-10;

  // D = distance between centers
  float D[3];
  for (int i = 0; i < 3; i++)
    D[i] = xB[i] - xA[i];

  // C = scalar product of axes
  float c[3][3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      c[i][j] = 0;
      for (int k = 0; k < 3; k++)
        c[i][j] = c[i][j] + vA[3*i+k]*vB[3*j+k];
    }
  }

  // separating axes based on the faces of box A
  for (int i = 0; i < 3; i++)
  {
    float r0 = dxA[i];
    float r1 = 0;
    float r = 0;
    for (int j = 0; j < 3; j++)
    {
      r1 += dxB[j] * fabs(c[i][j]);
      r += fabs(vA[3*i+j]) * D[j];
    }

    if (r > (r0+r1+eps))
      return 0;
  }

  // separating axes based on the faces of box B
  for (int i = 0; i < 3; i++)
  {
    float r1 = dxB[i];
    float r0 = 0;
    float r = 0;
    for (int j = 0; j < 3; j++)
    {
      r0 += dxA[j] * fabs(c[j][i]);
      r += fabs(vB[3*i+j]) * D[j];
    }
    if (r > (r0+r1+eps)) return 0;
  }

  // cross products
  for (int i = 0; i < 3; i++)
  {
    int i1 = (i+1)%3;
    int i2 = (i+2)%3;
    for (int j = 0; j < 3; j++)
    {
      int j1 = (j+1)%3;
      int j2 = (j+2)%3;

      float r0 = dxA[i1] * fabs(c[i2][j]) + dxA[i2] * fabs(c[i1][j]);
      float r1 = dxB[j1] * fabs(c[i][j2]) + dxB[j2] * fabs(c[i][j1]);

      float d2 = 0;
      float d1 = 0;
      for (int k = 0; k < 3; k++)
      {
        d2 += vA[3*i2+k]*D[k];
        d1 += vA[3*i1+k]*D[k];
      }

      float r = fabs( c[i1][j]*d2 - c[i2][j]*d1 );

      if (r > (r0+r1+eps)) {
        return 0;
      }
    }
  }

  // return 1 if no separation can be found [intersection]
  return 1;
}

/* From Demmel 1997, Applied Numerical Linear Algebra, pp. 232-235 */

//! Specific to a 3x3 matrix [used to get oriented bounding box axes]
__device__ __forceinline__
void jacobi_rotation(float A[3][3], float* __restrict__ J, int j, int k)
{
  if (abs(A[j][k]) > 1e-10)
  {
    float tau = (A[j][j] - A[k][k]) / (2*A[j][k]);
    float t = sign_d(tau) / (abs(tau) + sqrt(1+tau*tau));
    float c = 1. / (sqrt(1+t*t));
    float s = c*t;

    // Temporary matrix for matrix-matrix multiplication
    float B[6];

    // Apply R^T from left - rotates rows j,k by theta
    for (int d = 0; d < 3; d++)
    {
      B[d]   =  c*A[d][j] + s*A[d][k];
      B[d+3] = -s*A[d][j] + c*A[d][k];
    }

    // Update A to be result of R^T*A (Row i not changed)
    for (int d = 0; d < 3; d++)
    {
      A[j][d] = B[d];
      A[k][d] = B[d+3];
    }

    // Copy columns j,k of A to B
    for (int d = 0; d < 3; d++)
    {
      B[d]   = A[d][j];
      B[d+3] = A[d][k];
    }

    // Apply R from right - rotates cols j,k by theta
    for (int d = 0; d < 3; d++)
    {
      A[d][j] =  c*B[d] + s*B[d+3];
      A[d][k] = -s*B[d] + c*B[d+3];
    }

    // Copy columns j,k of J into tmp storage for multiplication
    for (int d = 0; d < 3; d++)
    {
      B[d]   = J[3*d+j];
      B[d+3] = J[3*d+k];
    }

    // Update J [eventual eigenvectors] [J = J*R]
    for (int d = 0; d < 3; d++)
    {
      J[3*d+j] =  c*B[d] + s*B[d+3];
      J[3*d+k] = -s*B[d] + c*B[d+3];
    }
  }
}

__device__ __forceinline__
float off_norm(float A[3][3])
{
  return sqrt(A[0][1]*A[0][1] + A[0][2]*A[0][2] + A[1][2]*A[1][2]);
}

//! Get the eigenvectors of a 3x3 symmetric matrix [covariance matrix]
__device__ __forceinline__
void get_eigenvectors(float mat[3][3], float* __restrict__ evecs)
{
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < i; j++)
      evecs[3*i+j] = 0;

    evecs[3*i+i] = 1;

    for (int j = i+1; j < 3; j++)
      evecs[3*i+j] = 0;
  }

  // 4 steps virtually guarantees good results
  for (int i = 0; i < 4; i++)
  {
    jacobi_rotation(mat, evecs, 0, 1);
    jacobi_rotation(mat, evecs, 0, 2);
    jacobi_rotation(mat, evecs, 1, 2);
  }
}

/*! Get the 3x3 covariance matrix for a set of points
 *  WARNING: upon return 'pts' will be shifted to place centroid at origin */
__device__ __forceinline__
void get_covariance(float* __restrict__ pts, int npts, float xc[3], float cov[3][3])
{
  // Get the centroid of all points
  xc[0] = xc[1] = xc[2] = 0.f;

  for (int i = 0; i < npts; i++)
    for (int d = 0; d < 3; d++)
      xc[d] += pts[3*i+d];

  for (int d = 0; d < 3; d++)
    xc[d] /= (float)npts;

  // Shift the centroid to the origin
  for (int i = 0; i < npts; i++)
    for (int d = 0; d < 3; d++)
      pts[3*i+d] -= xc[d];

  // Compute the covarience [just the upper-triangular terms]
  for (int d1 = 0; d1 < 3; d1++)
  {
    for (int d2 = d1; d2 < 3; d2++)
    {
      cov[d1][d2] = 0.f;
      for (int i = 0; i < npts; i++)
        cov[d1][d2] += pts[3*i+d1]*pts[3*i+d2];
      cov[d1][d2] /= (float)npts;
    }
  }

  // Apply symmetry
  for (int d1 = 0; d1 < 3; d1++)
    for (int d2 = d1; d2 < 3; d2++)
      cov[d2][d1] = cov[d1][d2];
}

/*! Get the oriented bounding box of a set of points using the eigenvectors
 *  of the covariance matrix as the axis of the box
 *  First 9 values of obb are the new axes; next 6 are transformed bbox */
__device__ __forceinline__
void getOBB(float* __restrict__ pts, int npts, float* obb)
{
  float cov[3][3], xc[3], axes[9];
  get_covariance(pts, npts, xc, cov);

  get_eigenvectors(cov, axes);

  for (int i = 0; i < 9; i++)
    obb[i] = axes[i];

  for (int i = 0; i < 3; i++)
  {
    obb[9+i]   =  BIG_DOUBLE;
    obb[9+i+3] = -BIG_DOUBLE;
  }

  for (int i = 0; i < npts; i++)
  {
    float pt[3] = {0.f, 0.f, 0.f};
    for (int j = 0; j < 3; j++)
      for (int k = 0; k < 3; k++)
        pt[j] += obb[3*k+j] * (pts[3*i+k]+xc[k]);

    for (int d = 0; d < 3; d++)
    {
      obb[9+d]   = fminf(obb[9+d],   pt[d]);
      obb[9+d+3] = fmaxf(obb[9+d+3], pt[d]);
    }
  }
}

/* ---------------------------- Direct Cut Method Functions --------------------------- */

static
__device__
float lineSegmentDistance(float *p1, float *p2, float *p3, float *p4, float *dx)
{
  // Get the line equations
  const float U[3] = {p2[0]-p1[0], p2[1]-p1[1], p2[2]-p1[2]};
  const float V[3] = {p4[0]-p3[0], p4[1]-p3[1], p4[2]-p3[2]};
  const float W[3] = {p1[0]-p3[0], p1[1]-p3[1], p1[2]-p3[2]};
  const float uu = U[0]*U[0] + U[1]*U[1] + U[2]*U[2];
  const float vv = V[0]*V[0] + V[1]*V[1] + V[2]*V[2];
  const float uv = U[0]*V[0] + U[1]*V[1] + U[2]*V[2];

  const float uw = U[0]*W[0] + U[1]*W[1] + U[2]*W[2];
  const float vw = V[0]*W[0] + V[1]*W[1] + V[2]*W[2];

  const float den = uu*vv - uv*uv;

  // NOTE: not finding exact minimum distance between the line segments in all
  // cases; plenty close enough for our purposes
  // (see http://geomalgorithms.com/a07-_distance.html for full algo)

  // Calculate line parameters (if nearly parallel, set one & calculate other)
  float s = (den < 1e-7f) ? 0.0f : (uv*vw - vv*uw) / den;
  float t = (den < 1e-7f) ? uw / uv: (uu*vw - uv*uw) / den;

  s = fminf(fmaxf(s, 0.f), 1.f);
  t = fminf(fmaxf(t, 0.f), 1.f);

  // vec = closest distance from segment 1 to segment 2
  for (int i = 0; i < 3; i++)
    dx[i] = t*V[i] - s*U[i] - W[i];

  float dist = 0.f;
  for (int i = 0; i < 3; i++)
    dist += dx[i]*dx[i];

  return sqrt(dist);
}

/*! Modified Moller triangle-triangle intersection algorithm
 *  Determines if triangles intersect, or returns an approximate minimum
 *  distance between them otherwise
 *  Also returns vector of minimum distance from T1 to T2 */
static
__device__
float triTriDistanceVec(float* __restrict__ T1, float* __restrict__ T2,
    float* __restrict__ minVec, float tol)
{
  float dist = 1e15f;
  float vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      float D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        for (int d = 0; d < 3; d++)
          minVec[d] = vec[d];
        dist = D;
      }
    }
  }

  // Pointers to points
  const float* V01 = T1;
  const float* V11 = T1+3;
  const float* V21 = T1+6;

  const float* V02 = T2;
  const float* V12 = T2+3;
  const float* V22 = T2+6;

  float N1[3], N2[3];

  // Plane for Triangle 1
  CROSS4(V11,V01, V21,V01, N1);

  float norm = NORM(N1);

  // Plane for Triangle 2
  for (int d = 0; d < 3; d++)
    N1[d] /= norm;

  float d1 = -DOT(N1,V01);

  CROSS4(V12,V02, V22,V02, N2);

  norm = NORM(N2);

  for (int d = 0; d < 3; d++)
    N2[d] /= norm;

  float d2 = -DOT(N2,V02);

  // Signed distances of T1's vertices to T2's plane
  float d01 = DOT(N2,V01) + d2;
  float d11 = DOT(N2,V11) + d2;
  float d21 = DOT(N2,V21) + d2;

  float d02 = DOT(N1,V02) + d1;
  float d12 = DOT(N1,V12) + d1;
  float d22 = DOT(N1,V22) + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-10) ? 0 : d01;
  d11 = (fabs(d11) < 1e-10) ? 0 : d11;
  d21 = (fabs(d21) < 1e-10) ? 0 : d21;

  d02 = (fabs(d02) < 1e-10) ? 0 : d02;
  d12 = (fabs(d12) < 1e-10) ? 0 : d12;
  d22 = (fabs(d22) < 1e-10) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    // Approximately coplanar; check if one triangle is inside the other /

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && DOTCROSS4(N2, V12,V02, V01,V02) > 0;
    inside = inside && DOTCROSS4(N2, V02,V22, V01,V22) > 0;
    inside = inside && DOTCROSS4(N2, V22,V12, V01,V12) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && DOTCROSS4(N1, V11,V01, V02,V01) > 0;
    inside = inside && DOTCROSS4(N1, V01,V21, V02,V21) > 0;
    inside = inside && DOTCROSS4(N1, V21,V11, V02,V11) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      float P01[3];
      for (int d = 0; d < 3; d++)
        P01[d] = V01[d] - N2[d]*d01;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P01,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P01,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P01,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d01; // Vector from T1 to T2
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      float P11[3];
      for (int d = 0; d < 3; d++)
        P11[d] = V11[d] - N2[d]*d11;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P11,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P11,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P11,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d11;
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      float P21[3];
      for (int d = 0; d < 3; d++)
        P21[d] = V21[d] - N2[d]*d21;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P21,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P21,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P21,V12) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = -N2[i]*d21;
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      float P02[3];
      for (int d = 0; d < 3; d++)
        P02[d] = V02[d] - N1[d]*d02;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P02,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P02,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P02,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d02;
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      float P12[3];
      for (int d = 0; d < 3; d++)
        P12[d] = V12[d] - N1[d]*d12;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P12,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P12,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P12,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d12;
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      float P22[3];
      for (int d = 0; d < 3; d++)
        P22[d] = V22[d] - N1[d]*d22;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P22,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P22,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P22,V11) > 0;

      if (inside)
      {
        for (int i = 0; i < 3; i++)
          minVec[i] = N1[i]*d22;
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  float L[3];
  CROSS(N1, N2, L);
  norm = NORM(L);
  for (int d = 0; d < 3; d++)
    L[d] /= norm;

  float p0 = DOT(L,V01);
  float p1 = DOT(L,V11);
  float p2 = DOT(L,V21);

  float q0 = DOT(L,V02);
  float q1 = DOT(L,V12);
  float q2 = DOT(L,V22);

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  float s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  float t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-10f) ? 0 : s1;
  s2 = (fabs(s2) < 1e-10f) ? 0 : s2;
  t1 = (fabs(t1) < 1e-10f) ? 0 : t1;
  t2 = (fabs(t2) < 1e-10f) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    float dt = fminf(fabs(t1-s2), fabs(s1-t2));
    float dl = 0;
    for (int d = 0; d < 3; d++)
      dl += (dt*L[d])*(dt*L[d]);
    dl = sqrt(dl);

    if (dl < dist)
    {
      dist = dl;
      for (int i = 0; i < 3; i++)
        minVec[i] = sgn(t1-s2)*dt*L[i]; // Ensure vec is T1 -> T2
    }

    return dist;
  }

  return 0.f;
}

/*! Modified Moller triangle-triangle intersection algorithm
 *  Determines if triangles intersect, or returns an approximate minimum
 *  distance between them otherwise */
static
__device__
float triTriDistance(float* __restrict__ T1, float* __restrict__ T2, float tol)
{
  float dist = 1e15f;
  float vec[3];
  for (int i = 0; i < 3; i++)
  {
    for (int j = 0; j < 3; j++)
    {
      int i2 = (i+1) % 3;
      int j2 = (j+1) % 3;
      float D = lineSegmentDistance(&T1[3*i], &T1[3*i2], &T2[3*j], &T2[3*j2], vec);

      if (D < dist)
      {
        dist = D;
      }
    }
  }

  // Pointers to points
  const float* V01 = T1;
  const float* V11 = T1+3;
  const float* V21 = T1+6;

  const float* V02 = T2;
  const float* V12 = T2+3;
  const float* V22 = T2+6;

  float N1[3], N2[3];

  // Plane for Triangle 1
  CROSS4(V11,V01, V21,V01, N1);

  float norm = NORM(N1);

  // Plane for Triangle 2
  for (int d = 0; d < 3; d++)
    N1[d] /= norm;

  float d1 = -DOT(N1,V01);

  CROSS4(V12,V02, V22,V02, N2);

  norm = NORM(N2);

  for (int d = 0; d < 3; d++)
    N2[d] /= norm;

  float d2 = -DOT(N2,V02);

  // Signed distances of T1's vertices to T2's plane
  float d01 = DOT(N2,V01) + d2;
  float d11 = DOT(N2,V11) + d2;
  float d21 = DOT(N2,V21) + d2;

  float d02 = DOT(N1,V02) + d1;
  float d12 = DOT(N1,V12) + d1;
  float d22 = DOT(N1,V22) + d1;

  // Round values near 0 to 0
  d01 = (fabs(d01) < 1e-7f) ? 0 : d01;
  d11 = (fabs(d11) < 1e-7f) ? 0 : d11;
  d21 = (fabs(d21) < 1e-7f) ? 0 : d21;

  d02 = (fabs(d02) < 1e-7f) ? 0 : d02;
  d12 = (fabs(d12) < 1e-7f) ? 0 : d12;
  d22 = (fabs(d22) < 1e-7f) ? 0 : d22;

  if (fabs(d01) + fabs(d11) + fabs(d21) < 3*tol ||
      fabs(d02) + fabs(d12) + fabs(d22) < 3*tol)
  {
    // Approximately coplanar; check if one triangle is inside the other /

    // Check if a point in T1 is inside T2
    bool inside = true;
    inside = inside && DOTCROSS4(N2, V12,V02, V01,V02) > 0;
    inside = inside && DOTCROSS4(N2, V02,V22, V01,V22) > 0;
    inside = inside && DOTCROSS4(N2, V22,V12, V01,V12) > 0;

    if (inside) return 0.;

    // Check if a point in T2 is inside T1
    inside = true;
    inside = inside && DOTCROSS4(N1, V11,V01, V02,V01) > 0;
    inside = inside && DOTCROSS4(N1, V01,V21, V02,V21) > 0;
    inside = inside && DOTCROSS4(N1, V21,V11, V02,V11) > 0;

    if (inside) return 0.;
  }

  bool noTouch = false;

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d01) == sgn(d11) && sgn(d01) == sgn(d21)) // && fabs(d01) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d01) < dist)
    {
      float P01[3];
      for (int d = 0; d < 3; d++)
        P01[d] = V01[d] - N2[d]*d01;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P01,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P01,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P01,V12) > 0;

      if (inside)
      {
        dist = fabs(d01);
      }
    }

    if (fabs(d11) < dist)
    {
      float P11[3];
      for (int d = 0; d < 3; d++)
        P11[d] = V11[d] - N2[d]*d11;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P11,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P11,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P11,V12) > 0;

      if (inside)
      {
        dist = fabs(d11);
      }
    }

    if (fabs(d21) < dist)
    {
      float P21[3];
      for (int d = 0; d < 3; d++)
        P21[d] = V21[d] - N2[d]*d21;
      bool inside = true;
      inside = inside && DOTCROSS4(N2, V12,V02, P21,V02) > 0;
      inside = inside && DOTCROSS4(N2, V02,V22, P21,V22) > 0;
      inside = inside && DOTCROSS4(N2, V22,V12, P21,V12) > 0;

      if (inside)
      {
        dist = fabs(d21);
      }
    }
  }

  // Check for intersection with plane - one point should have opposite sign
  if (sgn(d02) == sgn(d12) && sgn(d02) == sgn(d22)) // && fabs(d02) > tol)
  {
    noTouch = true;

    // No intersection; check if projection of points provides closer distance
    if (fabs(d02) < dist)
    {
      float P02[3];
      for (int d = 0; d < 3; d++)
        P02[d] = V02[d] - N1[d]*d02;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P02,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P02,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P02,V11) > 0;

      if (inside)
      {
        dist = fabs(d02);
      }
    }

    if (fabs(d12) < dist)
    {
      float P12[3];
      for (int d = 0; d < 3; d++)
        P12[d] = V12[d] - N1[d]*d12;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P12,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P12,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P12,V11) > 0;

      if (inside)
      {
        dist = fabs(d12);
      }
    }

    if (fabs(d22) < dist)
    {
      float P22[3];
      for (int d = 0; d < 3; d++)
        P22[d] = V22[d] - N1[d]*d22;
      bool inside = true;
      inside = inside && DOTCROSS4(N1, V11,V01, P22,V01) > 0;
      inside = inside && DOTCROSS4(N1, V01,V21, P22,V21) > 0;
      inside = inside && DOTCROSS4(N1, V21,V11, P22,V11) > 0;

      if (inside)
      {
        dist = fabs(d22);
      }
    }
  }

  // No intersection; return result from edge intersections & plane projections
  if (noTouch)
    return dist;

  // Compute intersection line
  float L[3];
  CROSS(N1, N2, L);
  norm = NORM(L);
  for (int d = 0; d < 3; d++)
    L[d] /= norm;

  float p0 = DOT(L,V01);
  float p1 = DOT(L,V11);
  float p2 = DOT(L,V21);

  float q0 = DOT(L,V02);
  float q1 = DOT(L,V12);
  float q2 = DOT(L,V22);

  // Figure out which point of each triangle is opposite the other two
  int npt1 = (sgn(d01) != sgn(d11)) ? ( (sgn(d11) == sgn(d21)) ? 0 : 1 ) : 2;
  int npt2 = (sgn(d02) != sgn(d12)) ? ( (sgn(d12) == sgn(d22)) ? 0 : 1 ) : 2;

  float s1, s2;
  switch (npt1)
  {
    case 0:
      s1 = p1 + (p0-p1) * (d11 / (d11-d01));
      s2 = p2 + (p0-p2) * (d21 / (d21-d01));
      break;
    case 1:
      s1 = p0 + (p1-p0) * (d01 / (d01-d11));
      s2 = p2 + (p1-p2) * (d21 / (d21-d11));
      break;
    case 2:
      s1 = p0 + (p2-p0) * (d01 / (d01-d21));
      s2 = p1 + (p2-p1) * (d11 / (d11-d21));
      break;
  }

  float t1, t2;
  switch (npt2)
  {
    case 0:
      t1 = q1 + (q0-q1) * (d12 / (d12-d02));
      t2 = q2 + (q0-q2) * (d22 / (d22-d02));
      break;
    case 1:
      t1 = q0 + (q1-q0) * (d02 / (d02-d12));
      t2 = q2 + (q1-q2) * (d22 / (d22-d12));
      break;
    case 2:
      t1 = q0 + (q2-q0) * (d02 / (d02-d22));
      t2 = q1 + (q2-q1) * (d12 / (d12-d22));
      break;
  }

  s1 = (fabs(s1) < 1e-7f) ? 0 : s1;
  s2 = (fabs(s2) < 1e-7f) ? 0 : s2;
  t1 = (fabs(t1) < 1e-7f) ? 0 : t1;
  t2 = (fabs(t2) < 1e-7f) ? 0 : t2;

  if (s1 > s2)
    swap(s1,s2);

  if (t1 > t2)
    swap(t1,t2);

  if (s2 < t1 || t2 < s1)
  {
    // No overlap; return min of dt*L and minDist
    float dt = fminf(fabs(t1-s2), fabs(s1-t2));
    float dl = 0;
    for (int d = 0; d < 3; d++)
      dl += (dt*L[d])*(dt*L[d]);
    dl = sqrt(dl);

    if (dl < dist)
      dist = dl;

    return dist;
  }

  return 0.f;
}

static
__device__ __forceinline__
dPointf faceNormal(const float* xv)
{
  /* Assuming nodes of face ordered CCW such that right-hand rule gives
     * outward normal */

  // Triangle #1
  dPointf pt0 = dPointf(&xv[0]);
  dPointf pt1 = dPointf(&xv[3]);
  dPointf pt2 = dPointf(&xv[6]);
  dPointf norm1 = (pt1-pt0).cross(pt2-pt0);           // Face normal vector

  // Triangle #2
  pt1 = dPointf(&xv[9]);
  dPointf norm2 = (pt2-pt0).cross(pt1-pt0);

  // Average the two triangle's normals
  dPointf norm = 0.5*(norm1+norm2);

  return (norm / norm.norm());
}

//! Calculate the distance between a linear triangle and a curved face
template<int nSideF>
__device__
float intersectionCheckOne(dMeshBlock &mb, const float* __restrict__ fxv,
    float* __restrict__ minVec, float* TC)
{
  /* --- Prerequisites --- */

  const int sorderF = nSideF-1;

  float tol = 1e-7f;
  float TF[9];
  float minDist = BIG_DOUBLE;
  minVec[0] = minDist;
  minVec[1] = minDist;
  minVec[2] = minDist;

  /* Only 3 cases possible:
   * 1) Face entirely contained within element
   * 2) Face intersects with element's boundary
   * 3) Face and element do not intersect
   */

  for (int M = 0; M < sorderF; M++)
  {
    for (int N = 0; N < sorderF; N++)
    {
      int m0 = M + nSideF*N;
      int TriPtsF[2][3] = {{m0, m0+1, m0+nSideF+1}, {m0, m0+nSideF+1, m0+nSideF}};
      for (int m = 0; m < 2; m++)
        for (int n = 0; n < 3; n++)
          TriPtsF[m][n] = mb.ijk2gmsh_quad[TriPtsF[m][n]];

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          int ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        float vec[3];
        float dist = triTriDistanceVec(TF, TC, vec, tol);

        if (dist < tol)
          return 0.;

        if (dist < minDist)
        {
          for (int d = 0; d < 3; d++)
            minVec[d] = vec[d];
          minDist = dist;
        }
      }
    }
  }

  return minDist;
}

//! Calc distance from near half of a hex to a face, assuming both linear
__device__ __forceinline__
float intersectionCheckLinear(const float* __restrict__ fxv,
    const float* __restrict__ exv, char &cornerOut)
{
  /* --- Prerequisites --- */

  // NOTE: Gmsh ordering  |  btm,top,left,right,front,back
  const char TriPts[12][3] = {{0,1,2},{0,2,3},{4,6,5},{4,7,6},{0,3,7},{0,7,4},
                       {1,2,6},{1,6,5},{0,4,5},{0,5,1},{3,2,6},{3,7,6}};

  float tol = 1e-7f;
  float TC[9], TF[9];
  float minDist = 1e15f;

  float xcf[3];
  cuda_funcs::getCentroid<3,4>(fxv,xcf);

  // Find nearest corner of element to face; check only that half of element
  int corner = -1;
  for (int i = 0; i < 8; i++)
  {
    float dist = 0.f;
    for (int d = 0; d < 3; d++)
      dist += (exv[3*i+d] - xcf[d]) * (exv[3*i+d] - xcf[d]);

    if (dist < minDist)
    {
      minDist = dist;
      corner = i;
    }
  }

  cornerOut = corner;

  // Faces 0 or 1, 2 or 3, and 4 or 5 (btm or top, L or R, etc.)
  const char fList[3] = {(char)(corner / 4), (char) (((corner + 1)%4) / 2 + 2), (char)(((corner%4) / 2) + 4)};

  // 3) Check those faces of element for intersection with face
  for (int F = 0; F < 3; F++)
  {
    char f = fList[F];
    // Get triangles for the sub-hex of the larger curved hex
    for (int i = 2*f; i < 2*f+2; i++)
    {
      for (int p = 0; p < 3; p++)
      {
        char ipt = TriPts[i][p];
        for (int d = 0; d < 3; d++)
          TC[3*p+d] = exv[3*ipt+d];
      }

      const char TriPtsF[2][3] = {{0, 1, 3}, {1, 2, 3}};

      // Intersection check between element face tris & cutting-face tris
      for (int j = 0; j < 2; j++)
      {
        for (int p = 0; p < 3; p++)
        {
          char ipt = TriPtsF[j][p];
          for (int d = 0; d < 3; d++)
            TF[3*p+d] = fxv[3*ipt+d];
        }

        float dist = triTriDistance(TF, TC, tol);

        if (dist < tol)
          return 0.;

        minDist = (dist < minDist) ? dist : minDist;
      }
    }
  }

  return minDist;
}

//! Perform initial ele-face distance sorting using oriented bounding boxes
__global__
void cuttingPass0B(dvec<float> eleBbox, dvec<float> eleXC, dvec<float> faceXC,
    dvec<float> cutFaces, int nEles, int nFaces, int nvertf, dvec<int> filt_faces, dvec<float> outDist)
{
  const int IC = blockIdx.x * blockDim.x + threadIdx.x;
  const int F = blockIdx.y * blockDim.y + threadIdx.y;

  if (IC >= nEles || F >= nFaces) return;

  const int nDims = 3;

  const int ff = filt_faces[F];
  const int stride = nDims*nvertf;

  /// Alternatively: compute distance to ele bbox for all 4 corner points
  float pts[12];
  for (int i = 0; i < 4; i++)
  {
    float pt[3];
    for (int j = 0; j < 3; j++)
      pt[j] = cutFaces[ff*stride+3*i+j];

    for (int j = 0; j < 3; j++)
    {
      pts[3*i+j] = 0;
      for (int k = 0; k < 3; k++)
      {
        pts[3*i+j] += eleBbox[16*IC+3*k+j] * pt[k];
      }
    }
  }

  float bboxF[6];
  cuda_funcs::getBoundingBox<3,4>(pts, bboxF);

  float dist = 0;
  for (int i = 0; i < 3; i++)
    dist += (eleXC[3*IC+i]-faceXC[3*F+i])*(eleXC[3*IC+i]-faceXC[3*F+i]);
  dist = sqrt(dist);

  outDist[nEles*F+IC] = .01f*dist + cuda_funcs::boundingBoxDist<3>(bboxF, &eleBbox[16*IC+9]);
}

__global__
void sortFaces0(dvec<int> faceList, int nEles, int nFaces, dvec<float> distList,
    dvec<int> outFaces)
{
  const int IC = blockDim.x * blockIdx.x + threadIdx.x;

  if (IC >= nEles) return;

  float dists[NF1];
  int faces[NF1];

  for (int i = 0; i < NF1; i++)
  {
    dists[i] = BIG_FLOAT;
    faces[i] = -1;
  }

  for (int F = 0; F < nFaces; F++)
  {
    int ind = NF1-1;

    float Dist = distList[nEles*F+IC];
    if (Dist > dists[ind]) continue;

    dists[ind] = Dist;
    faces[ind] = faceList[F];
    while (ind > 0 && dists[ind] < dists[ind-1])
    {
      swap(dists[ind], dists[ind-1]);
      swap(faces[ind], faces[ind-1]);
      ind--;
    }
  }

  for (int i = 0; i < NF1; i++)
  {
    outFaces[nEles*i+IC] = faces[i];
  }
}

__global__
void cuttingPass1(dMeshBlock mb, dvec<int> filt_eles, int nEles,
    dvec<float> cutFaces, int nvertf, int nFaces, dvec<int> checkFaces,
    dvec<char> outCorner, dvec<float> outDist)
{
  const int IC = blockIdx.x * blockDim.x + threadIdx.x;
  const int F = blockIdx.y * blockDim.y + threadIdx.y;

  if (IC >= nEles || F >= nFaces) return;

  const int nDims = 3;

  const int ic = filt_eles[IC];  // Get filtered cell ID
  const int ff = checkFaces[nEles*F+IC];
  const int stride = nDims*nvertf;

  // Load up the cell nodes into an array
  float xv[8*nDims];
  for (int i = 0; i < 8*nDims; i++)
  {
    int d = i % 3;
    int v = i / 3;
    xv[i] = mb.coord[ic+mb.ncells*(d+nDims*v)]; /// NOTE: 'row-major' ZEFR layout
  }

  // Each thread will check against 1 face
  outDist[nEles*F+IC] = intersectionCheckLinear(&cutFaces[ff*stride], xv, outCorner[nEles*F+IC]);
}

__global__
void sortFaces(int nEles, int nFaces, dvec<float> distList,
    dvec<char> cornerList, dvec<int> outFaces, dvec<char> outCorners)
{
  const int IC = blockDim.x * blockIdx.x + threadIdx.x;

  if (IC >= nEles) return;

  float dists[NF2];
  int faces[NF2];
  char corners[NF2];

  for (int i = 0; i < NF2; i++)
  {
    dists[i] = BIG_FLOAT;
    faces[i] = -1;
    corners[i] = -1;
  }

  for (int F = 0; F < nFaces; F++)
  {
    int ind = NF2-1;

    float Dist = distList[nEles*F+IC];
    if (Dist > dists[ind]) continue;

    dists[ind] = Dist;
    faces[ind] = outFaces[nEles*F+IC];
    corners[ind] = cornerList[nEles*F+IC];
    while (ind > 0 && dists[ind] < dists[ind-1])
    {
      swap(dists[ind], dists[ind-1]);
      swap(faces[ind], faces[ind-1]);
      swap(corners[ind], corners[ind-1]);
      ind--;
    }
  }

  for (int i = 0; i < NF2; i++)
  {
    outFaces[nEles*i+IC] = faces[i];
    outCorners[nEles*i+IC] = corners[i];
  }
}

__global__
void getFaceCentroids(dvec<float> cutFaces, dvec<int> faceList, int nFaces,
    int nvertf, dvec<float> faceCentroid)
{
  int F = blockIdx.x * blockDim.x + threadIdx.x;

  if (F >= nFaces) return;

  int ff = faceList[F];

  cuda_funcs::getCentroid<3,4>(&cutFaces[3*nvertf*ff], &faceCentroid[3*F]);
}

__global__
void getElementCentroids(dMeshBlock mb, dvec<int> eleList, int nEles,
    dvec<float> eleCentroid)
{
  int IC = blockIdx.x * blockDim.x + threadIdx.x;

  if (IC >= nEles) return;

  int ic = eleList[IC];

  float xv[8*3];  // Only concerning ourselves with linear portion of ele
  for (int i = 0; i < 8; i++)
    for (int d = 0; d < 3; d++)
      xv[3*i+d] = mb.coord[ic+mb.ncells*(d+3*i)];

  cuda_funcs::getCentroid<3,8>(xv,&eleCentroid[3*IC]);
}

__global__
void getElementBoundingBoxes(dMeshBlock mb, dvec<int> eleList, int nEles,
    dvec<float> eleBbox)
{
  int IC = blockIdx.x * blockDim.x + threadIdx.x;

  if (IC >= nEles) return;

  int ic = eleList[IC];

  float xv[8*3];  // Only concerning ourselves with linear portion of ele
  for (int i = 0; i < 8; i++)
    for (int d = 0; d < 3; d++)
      xv[3*i+d] = mb.coord[ic+mb.ncells*(d+3*i)];

  cuda_funcs::getBoundingBox<3,8>(xv,&eleBbox[6*IC]); /// TODO: swap rows/cols in eleBbox for better coalesced access
}

__global__
void getElementOrientedBoundingBoxes(dMeshBlock mb, dvec<int> eleList, int nEles,
    dvec<float> eleBbox)
{
  int IC = blockIdx.x * blockDim.x + threadIdx.x;

  if (IC >= nEles) return;

  int ic = eleList[IC];

  float xv[8*3];  // Only concerning ourselves with linear portion of ele
  for (int i = 0; i < 8; i++)
    for (int d = 0; d < 3; d++)
      xv[3*i+d] = mb.coord[ic+mb.ncells*(d+3*i)];

  getOBB(xv, 8, &eleBbox[16*IC]);

  // Use last entry for 'href' [average length of oriented bounding box]
  eleBbox[16*IC+15] = ( (eleBbox[16*IC+12] - eleBbox[16*IC+9]) +
                        (eleBbox[16*IC+13] - eleBbox[16*IC+10]) +
                        (eleBbox[16*IC+14] - eleBbox[16*IC+11]) ) / 3.;
}

template<int nSideF>
__global__
void cuttingPass2(dMeshBlock mb, dvec<float> cutFaces, dvec<int> checkFaces,
    dvec<int> list, int nEles, int nSideC, dvec<float> eleBbox,
    dvec<char> corners, dvec<float> outDist, dvec<float> outVec)
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int FID = threadIdx.y;

  const int nDims = 3;
  const int nvertf = nSideF*nSideF;
  const int stride = nDims*nvertf;
  const int sOrderC = nSideC - 1;

  const int nQuadFace = sOrderC*sOrderC;
  const int nTriFace = 2*nQuadFace;
  const int nTri = 3*nTriFace;

  /* --- Get our specific element & sub-triangle of element --- */

  const int IC = tid / (3 * nTriFace);
  const int T = tid % (3 * nTriFace);

  const unsigned char q = (T % (nTriFace)) / 2;
  const unsigned char t = (T % (nTriFace)) % 2;
  char F = T / nTriFace;

  if (IC >= nEles) return;

  const int ic = list[IC];  // Get filtered cell ID
  const int ff = checkFaces[nEles*FID+IC];

  if (ff < 0)
    return;

  /* ---- Check against our reduced list of faces ---- */

  const float href = eleBbox[16*IC+15];
  const float dtol = .2*href;

  // Only checking half the element's faces; figure out which ones
  const char corner = corners[nEles*FID+IC];

  switch (F)
  {
    case 0:
      F = corner / 4;
      break;
    case 1:
      F = ((corner + 1)%4) / 2 + 2;
      break;
    case 2:
      F = (((corner%4) / 2) + 4);
      break;
  }

  /* ---- Get our specific triangle ---- */

  // NOTE: Structured ordering  |  btm,top,left,right,front,back
  const char TriPts[12][3] = {{0,1,3},{0,3,2},{4,7,5},{4,6,7},{0,2,6},{0,6,4},
    {1,3,7},{1,7,5},{0,4,5},{0,5,1},{2,3,7},{2,6,7}};

  int I, J, K;
  switch (F)
  {
    case 0: // Bottom
      I = q / sOrderC;
      J = q % sOrderC;
      K = 0;
      break;
    case 1: // Top
      I = q / sOrderC;
      J = q % sOrderC;
      K = sOrderC - 1;
      break;
    case 2: // Left
      I = 0;
      J = q / sOrderC;
      K = q % sOrderC;
      break;
    case 3: // Right
      I = sOrderC - 1;
      J = q / sOrderC;
      K = q % sOrderC;
      break;
    case 4: // Front
      I = q / sOrderC;
      J = 0;
      K = q % sOrderC;
      break;
    case 5: // Back
      I = q / sOrderC;
      J = sOrderC - 1;
      K = q % sOrderC;
      break;
  }

  int i0 = I+nSideC*(J+nSideC*K);
  int j0 = i0 + nSideC*nSideC;
  int lin2curv[8] = {i0, i0+1, i0+nSideC, i0+nSideC+1, j0, j0+1, j0+nSideC, j0+nSideC+1};
  for (int i = 0; i < 8; i++)
    lin2curv[i] = mb.ijk2gmsh[lin2curv[i]];

  float TC[9];
  for (int p = 0; p < 3; p++)
  {
    int ipt = lin2curv[TriPts[2*F+t][p]];
    for (int d = 0; d < 3; d++)
      TC[3*p+d] = mb.coord[ic+mb.ncells*(d+nDims*ipt)]; // NOTE: 'row-major' ZEFR layout
  }

  /* Find distance from face to cell
   * NOTE: ignoring case of face entirely inside cell, since any valid grid
   * will also have a different face which intersects its boundary */
  dPointf vec;
  float myDist = intersectionCheckOne<nSideF>(mb, &cutFaces[ff*stride], &vec[0], TC);
  vec /= vec.norm();

  if (myDist < dtol) // They intersect
  {
    myDist = 0.;
  }

  // Write out results to global memory for future reduction
  outDist[T+nTri*(IC+nEles*FID)] = myDist;
  for (int i = 0; i < 3; i++)
  {
    outVec[T+nTri*(IC+nEles*(FID+NF2*i))] = vec[i];
  }
}

__global__
void getMinDist(dvec<float> dists, dvec<float> vecs, int nEles, int nTri)
{
  const int IC = blockDim.x * blockIdx.x + threadIdx.x;
  const int F = threadIdx.y;

  if (IC >= nEles) return;

  // Find minimum tri-face distance for this face/element
  float minDist = BIG_FLOAT;
  float myVec[3] = {0.0f};
  for (int i = 0; i < nTri; i++)
  {
    float dist = dists[i+nTri*(IC+nEles*F)];

    if (dist < minDist)
    {
      minDist = dist;
      for (int d = 0; d < 3; d++)
        myVec[d] = vecs[i+nTri*(IC+nEles*(F+NF2*d))];
    }
  }

  // NOTE: Assuming NF2 always <= nTri
  dists[F+nTri*IC] = minDist;
  for (int d = 0; d < 3; d++)
    vecs[F+nTri*(IC+nEles*NF2*d)] = myVec[d];
}

__global__
void getFinalFlag(dvec<int> eleList, dvec<int> checkFaces,
    dvec<float> cutFaces, dvec<float> eleBbox, int nEles, int nvertf, int nTri,
    dvec<int> cutFlag, dvec<float> dists, dvec<float> vecs, int cutType)
{
  const int IC = blockDim.x * blockIdx.x + threadIdx.x;

  if (IC >= nEles) return;

  const int ic = eleList[IC];

  const float dtol = .1*eleBbox[16*IC+15];

  // Find nearest face distance for this element

  int nMin = 0;
  char myFlag = DC_UNASSIGNED;
  float minDist = BIG_FLOAT;
  float myDot = 0.f;
  dPointf myNorm;

  for (int F = 0; F < NF2; F++)
  {
    int ff = checkFaces[nEles*F+IC];
    dPointf norm = faceNormal(&cutFaces[ff*nvertf*3]);
    if (cutType == 0) norm *= -1;

    const float dist = dists[F+nTri*IC];

    dPointf vec;
    for (int d = 0; d < 3; d++)
      vec[d] = vecs[F+nTri*(IC+nEles*NF2*d)];

    const float dot = norm*vec;

    if (dist < dtol) // They intersect
    {
      myFlag = DC_CUT;
      minDist = 0.f;
    }
    else if (myFlag == DC_UNASSIGNED || (dist < (minDist - .02f*dtol) && fabs(dot) > 0.5f*fabs(myDot)))
    {
      // Unflagged cell, or have a closer face to use
      minDist = dist;
      myNorm = norm;
      myDot = myNorm*vec;

      nMin = 1;

      if (myDot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
    else if (fabs(dist - minDist) <= .02f*dtol && minDist > 0.f && fabs(dot) > 0.5f*fabs(myDot))
    {
      if (fabs(dot) > fabs(myDot) - .01f)
      {
        if (fabs(dot) > fabs(myDot))
        {
          myNorm = norm; // swap to better normal
          myDot = dot;
        }
      }
      else if (sgn(dot) != sgn(myDot))
      {
        // Approx. same dist. to two faces; avg. their normals to decide
        minDist = dist;
        for (int d = 0; d < 3; d++)
          myNorm[d] = (nMin*myNorm[d] + norm[d]) / (nMin + 1.f);
        nMin++;

        myDot = myNorm*vec;
      }

      if (myDot < 0)
        myFlag = DC_HOLE; // outwards normal = inside cutting surface
      else
        myFlag = DC_NORMAL;
    }
  }

  if (myFlag == DC_CUT)
    myFlag = (cutType != 0) ? DC_HOLE : DC_NORMAL;

  // Write out final result
  cutFlag[ic] = myFlag;
}

/*! Remove all elements which do not intersect with cut group's bbox from
 *  consideration (obviously do not intersect) */
template<int nvert>
__global__
void filterElements(dMeshBlock mb, dvec<double> cut_bbox, dvec<int> filt,
    dvec<int> cutFlag, dvec<int> nfilt, dvec<float> bboxOut, int cutType)
{
  const unsigned int ic = blockIdx.x * blockDim.x + threadIdx.x;

  // Initialize nfilt to 0; will be atomically added to at end
  if (ic == 0)
  {
    nfilt[0] = 0;
    for (int d = 0; d < 3; d++)
    {
      bboxOut[d]   =  1.e10f;
      bboxOut[d+3] = -1.e10f;
    }
  }

  __shared__ float bboxF[6];

  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    bboxF[i] = (float)cut_bbox[i];

  __syncthreads();

  if (ic >= mb.ncells) return;

  // Set all cell flags initially to DC_NORMAL (filtered cells will remain 'NORMAL')
  cutFlag[ic] = DC_NORMAL;

  // Use hole map block size as ref. length [times ~sqrt(3)]
  float href = 0.f;
  for (int i = 0; i < 3; i++)
    href = fmaxf(href, 2.f*(mb.hm_extents[i+3]-mb.hm_extents[i]) / mb.hm_nx[i]);
  //if(ic == 0)
  //for(int i=0;i<nvert;i++) {
  //  printf(" %d\n", mb.ijk2gmsh[i]);
  //}
  // Get element nodes
  float xv[nvert*3];
  for (int i = 0; i < nvert; i++) {
    for (int d = 0; d < 3; d++) {
      xv[3*i+d] = (float)mb.coord[ic+mb.ncells*(d+3*i)];
      //int gnd = mb.ijk2gmsh[i];
      //xv[3*gnd+d] = (float)mb.coord[ic+mb.ncells*(d+3*i)];
    }
  }
  // Get element bounding box
  float bboxC[6], xc[3];
  cuda_funcs::getBoundingBox<3,nvert>(xv, bboxC);

  bool tag1 = false; // Inside hole
  bool tag2 = false; // Near boundary of hole

  if (cutType == 0)
    tag1 = true;


  // Check the hole map status of all nodes
  for (int i = 0; i < 8; i++)
  {
    for (int d = 0; d < 3; d++)
      xc[d] = xv[3*i+d];
    
    //if(ic == 0) printf("x y z %lf %lf %lf\n", xc[0], xc[1], xc[2]);
    if (mb.rrot) // Transform xc to hole map's coordinate system
    {
      double x2[3] = {0.,0.,0.};
      for (int d1 = 0; d1 < 3; d1++)
        for (int d2 = 0; d2 < 3; d2++) {
          x2[d1] += mb.Rmat[d1+3*d2]*(xc[d2]-mb.offset[d2]); //! TODO: include Rmat from other grid
        }
      
      for (int d = 0; d < 3; d++)
        xc[d] = x2[d];
    }

    char tag = cuda_funcs::checkHoleMap<float>(xc, mb.hm_sam.data(), mb.hm_nx.data(), mb.hm_extents.data());

    if (cutType == 1) // Wall boundary - if any nodes have tag == 1, set as hole
      tag1 = tag1 || (tag == 1);
    else      // Overset boundary - only remove elements if entirely in hole map
      tag1 = tag1 && (tag == 1);

    // Otherwise, if any nodes have non-zero tag, keep around for distance calc
    tag2 = tag2 || (tag != 0);
  }

  bool checkH = !tag1;
  bool checkB = cuda_funcs::boundingBoxCheck<3>(bboxC, bboxF, href);

  // If filtering element due to being completely inside hole region, tag as hole
  if (tag1)
    cutFlag[ic] = DC_HOLE;

  if ( checkH && (checkB || tag2) ) // Not a hole, but near the boundary
  {
    int ind = atomicAggInc(&nfilt[0]);
    filt[ind] = ic;
    for (int d = 0; d < 3; d++)
    {
      atomicMinf(&bboxOut[d], bboxC[d]);
      atomicMaxf(&bboxOut[d+3], bboxC[d+3]);
    }
  }
}

/*! Remove all cutting faces which do not intersect this rank's reduced bbox
 *  from consideration (obviously do not intersect) */
template<int nvertf>
__global__
void filterFaces(dMeshBlock mb, dvec<float> ele_bbox, int nCut,
    dvec<float> cutFaces, dvec<int> filt, dvec<int> nfilt)
{
  const unsigned int ff = blockIdx.x * blockDim.x + threadIdx.x;

  // Initialize nfilt to 0; will be atomically added to at end
  if (ff == 0)
    nfilt[1] = 0;

  __shared__ float bboxE[6]; // Global bounding box of all filtered elements

  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    bboxE[i] = ele_bbox[i];

  __syncthreads();

  if (ff >= nCut) return;

  // Use hole map block size as ref. length [times ~sqrt(3)]
  float href = 0.f;
  for (int i = 0; i < 3; i++)
    href = fmaxf(href, 2.f*(mb.hm_extents[i+3]-mb.hm_extents[i]) / mb.hm_nx[i]);

  // Get face nodes
  float fxv[nvertf*3];
  for (int i = 0; i < nvertf; i++)
    for (int d = 0; d < 3; d++)
      fxv[3*i+d] = (float)cutFaces[(ff*nvertf+i)*3+d];

  // Get face bounding box
  float bboxF[6];
  cuda_funcs::getBoundingBox<3,nvertf>(fxv, bboxF);

  bool checkB = cuda_funcs::boundingBoxCheck<3>(bboxF, bboxE, href);

  if (checkB)
  {
    int ind = atomicAggInc(&nfilt[1]);
    filt[ind] = ff;
  }
}

// for debugging purpose
__global__
void print_device_data(int ncells, dvec<int> cutFlag ) {

  const unsigned int ic = blockIdx.x * blockDim.x + threadIdx.x;
  if(ic>= ncells) return;
  printf("cutFlag is %d\n", cutFlag[ic]);
}

void dMeshBlock::directCut(double* cutFaces_h, int nCut, int nvertf, double *cutBbox_h, int* cutFlag, int cutType)
{
  // Setup cutMap
  
  ///////////////////////////////////////////////////////
  // for debugging purpose
  /*
  printf("nCut is %d nvertf is %d\n", nCut, nvertf);
  // printf the face to check ordering of the nodes
  FILE* fp = fopen("check_face_nodes.dat","w");

  for (int j=0;j<nCut;j++) {
    for(int i=0;i<nvertf;i++) {
      int dim =3;
      fprintf(fp,"%lf %lf %lf %d\n",
                     cutFaces_h[j*nvertf*dim+i*dim+0],
                     cutFaces_h[j*nvertf*dim+i*dim+1],
                     cutFaces_h[j*nvertf*dim+i*dim+2],
                     i
                     );
    }
  }
  fclose(fp);
  */
  /////////////////////////////////////////////////////////
  
  cutFlag_d.resize(ncells);
  filt_eles.resize(ncells);
  filt_faces.resize(nCut);

  std::vector<float> cutFaces_hf(nCut*nvertf*nDims);
  for (int i = 0; i < nvertf*nDims*nCut; i++)
    cutFaces_hf[i] = (float)cutFaces_h[i];

  dvec<float> cutFaces;
  cutFaces.assign(cutFaces_hf.data(), nCut*nvertf*nDims);

  dvec<double> cutBbox_d;
  cutBbox_d.assign(cutBbox_h, 2*nDims);

  if (ijk2gmsh_quad.size() != nvertf)
  {
    auto ijk2gmsh_quad_h = tg_funcs::structured_to_gmsh_quad(nvertf);
    ijk2gmsh_quad.assign(ijk2gmsh_quad_h.data(), nvertf);

    //for(auto && i : ijk2gmsh_quad_h) {
    //  printf(" %d ", i);
    //}
    //printf("\n");
  }

  /* Filter elements based upon cutting-surface bounding box & Cartesian approx. rep. */

  hvec<int> nfilt_h;
  dvec<int> nfilt_d;
  nfilt_h.resize(2);
  nfilt_h[0] = 0;  nfilt_h[1] = 0;
  nfilt_d.assign(nfilt_h.data(), nfilt_h.size());

  ele_bbox.resize(6);

  int threads = 128;
  int blocks = (ncells + threads - 1) / threads;
  
  ///////////////// for debugging purpose ////////////////////
  //print_device_data<<<blocks, threads>>>(ncells, cutFlag_d);
  ///////////////// for debugging purpose ////////////////////

  switch(nvert)
  {
    case 8:
      filterElements<8><<<blocks, threads, 6*sizeof(float)>>>(*this, cutBbox_d, filt_eles, cutFlag_d, nfilt_d, ele_bbox, cutType);
      break;
    case 27:
      filterElements<27><<<blocks, threads, 6*sizeof(float)>>>(*this, cutBbox_d, filt_eles, cutFlag_d, nfilt_d, ele_bbox, cutType);
      break;
    case 64:
      filterElements<64><<<blocks, threads, 6*sizeof(float)>>>(*this, cutBbox_d, filt_eles, cutFlag_d, nfilt_d, ele_bbox, cutType);
      break;
    default:
      printf("nvert = %d\n",nvert);
      ThrowException("nvert case not implemented for filterElements on device");
  }
  check_error();

  if (nCut == 0)
  {
    cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);

    cutFaces.free_data();
    cutBbox_d.free_data();
    nfilt_h.free_data();
    nfilt_d.free_data();

    return;
  }

  //cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);
  //return;
  /* Filter cutting faces by intersection with the filtered elements' bounding box */

  blocks = (nCut + threads - 1) / threads;

  switch(nvertf)
  {
    case 4:
      filterFaces<4><<<blocks, threads, 6*sizeof(float)>>>(*this, ele_bbox, nCut, cutFaces, filt_faces, nfilt_d);
      break;
    case 9:
      filterFaces<9><<<blocks, threads, 6*sizeof(float)>>>(*this, ele_bbox, nCut, cutFaces, filt_faces, nfilt_d);
      break;
    case 16:
      filterFaces<16><<<blocks, threads, 6*sizeof(float)>>>(*this, ele_bbox, nCut, cutFaces, filt_faces, nfilt_d);
      break;
    default:
      printf("nvertf = %d\n",nvertf);
      ThrowException("nvertf case not implemented for filterFaces on device");
  }
  check_error();

  ///////////////// for debugging purpose ////////////////////
  //print_device_data<<<blocks, threads>>>(ncells, cutFlag_d);
  ///////////////// for debugging purpose ////////////////////

  nfilt_h.assign(nfilt_d.data(), 2);
  int nfiltC = nfilt_h[0];
  int nfiltF = nfilt_h[1];

  /* Perform the Direct Cut algorithm on the filtered list of elements & faces */

  int nSideC = std::cbrt(nvert);
  int nTri = 3*2*(nSideC-1)*(nSideC-1);

  if (nTri < NF2) ThrowException("Incompatible nTri / NF2!");

  int nCheck1 = min(NF1, nfiltF);

  dvec<float> eleBbox, cfDist, cfNorm, cfVec;
  dvec<char> cfCorner;
  cfDist.resize(nfiltC*nfiltF);
  cfCorner.resize(nfiltC*NF1);

  dvec<int> checkFaces;
  dvec<char> corners;
  checkFaces.resize(NF1*nfiltC);
  corners.resize(NF2*nfiltC);

  dvec<float> eleXC, faceXC;

  if (nfiltC > 0 && nfiltF > 0)
  {
    /* Pass 0: 'Coarse-Grained' Check using Bounding Boxes
     * Pass 1: 'Medium-Grained' Check using Linear Parts of Elements/Faces
     * Pass 2: 'Finest-Grained' Direct Cut Check
     */

    // Tell CUDA to offer more space for registers over shared memory
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(getElementBoundingBoxes), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(sortFaces0), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(sortFaces), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuttingPass0B), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuttingPass1), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuttingPass2<2>), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuttingPass2<3>), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuttingPass2<4>), hipFuncCachePreferL1);

    eleBbox.resize(16*nfiltC);

    eleXC.resize(nfiltC*3);
    faceXC.resize(nfiltF*3);

    int ThreadsB = 128;
    int BlocksB = (nfiltC + ThreadsB - 1) / ThreadsB;

    getElementOrientedBoundingBoxes<<<BlocksB,ThreadsB>>>(*this, filt_eles, nfiltC, eleBbox);

    getElementCentroids<<<BlocksB,ThreadsB>>>(*this, filt_eles, nfiltC, eleXC);
    int ThreadsF = 128;
    int BlocksF = (nfiltF + ThreadsF - 1) / ThreadsF;

    getFaceCentroids<<<BlocksF,ThreadsF>>>(cutFaces, filt_faces, nfiltF, nvertf, faceXC);
    check_error();

    // Have each filtered element calculate a rough distance to each filtered face

    dim3 Threads0(4,32);
    dim3 Blocks0( (nfiltC + Threads0.x - 1) / Threads0.x,
                  (nfiltF + Threads0.y - 1) / Threads0.y );

    cuttingPass0B<<<Blocks0,Threads0>>>(eleBbox, eleXC, faceXC, cutFaces, nfiltC, nfiltF, nvertf, filt_faces, cfDist);

    int ThreadsS0 = 128;
    int BlocksS0 = (nfiltC + ThreadsS0 - 1) / ThreadsS0;

    sortFaces0<<<BlocksS0, ThreadsS0>>>(filt_faces, nfiltC, nfiltF, cfDist, checkFaces);
    check_error();

    // Have each filtered element calculate a rough distance to the reduced face list

    dim3 Threads1(32,4);
    dim3 Blocks1( (nfiltC + Threads1.x - 1) / Threads1.x,
                  (nCheck1 + Threads1.y - 1) / Threads1.y );

    cuttingPass1<<<Blocks1,Threads1>>>(*this, filt_eles, nfiltC, cutFaces,
        nvertf, nCheck1, checkFaces, cfCorner, cfDist);

    // Sort the distance lists for each element [down to NF2 faces to check in detail]

    int ThreadsS1 = 128;
    int BlocksS1 = (nfiltC + ThreadsS1 - 1) / ThreadsS1;

    sortFaces<<<BlocksS1, ThreadsS1>>>(nfiltC, nCheck1, cfDist, cfCorner, checkFaces, corners);
    check_error();

    cfNorm.resize(3*nfiltC*nTri*NF2);
    cfVec.resize(3*nfiltC*nTri*NF2);
    cfDist.resize(nfiltC*nTri*NF2);

    dim3 t3(32,NF2);
    int b3 = (nfiltC*nTri + t3.x - 1) / t3.x;

    switch(nvertf)
    {
      case 4:
        cuttingPass2<2><<<b3, t3>>>(*this, cutFaces, checkFaces, filt_eles,
            nfiltC, nSideC, eleBbox, corners, cfDist, cfVec);
        break;

      case 9:
        cuttingPass2<3><<<b3, t3>>>(*this, cutFaces, checkFaces, filt_eles, nfiltC, nSideC, eleBbox, corners, cfDist, cfVec);
        break;

      case 16:
        cuttingPass2<4><<<b3, t3>>>(*this, cutFaces, checkFaces, filt_eles, nfiltC, nSideC, eleBbox, corners, cfDist, cfVec);
        break;

      default:
        printf("nvertFace = %d, nCut = %d\n",nvertf,nCut);
        ThrowException("nvertFace case not implemented for directCut on device");
    }
    check_error();

    int BlocksM = (nfiltC + t3.x - 1) / t3.x;
    getMinDist<<<BlocksM, t3>>>(cfDist, cfVec, nfiltC, nTri);

    getFinalFlag<<<BlocksM, 128>>>(filt_eles, checkFaces, cutFaces, eleBbox,
        nfiltC, nvertf, nTri, cutFlag_d, cfDist, cfVec, cutType);
    check_error();
  }

  cuda_copy_d2h(cutFlag_d.data(), cutFlag, ncells);

  ///////////////// for debugging purpose ////////////////////
  //print_device_data<<<blocks, threads>>>(ncells, cutFlag_d);
  ///////////////// for debugging purpose ////////////////////
  // Free all data allocated in this function

  cfDist.free_data();
  cfCorner.free_data();
  cfNorm.free_data();
  cfVec.free_data();
  eleBbox.free_data();
  eleXC.free_data();
  faceXC.free_data();

  nfilt_d.free_data();
  nfilt_h.free_data();

  corners.free_data();
  checkFaces.free_data();

  cutFaces.free_data();
  cutBbox_d.free_data();
}
